#include <hip/hip_runtime.h>
#include "minimap.h"
#include "mmpriv.h"
#include "kalloc.h"
#include "chain_cuda_kernel.h"

extern "C" {

void mm_chain_dp_fill_cuda(int max_dist_x, int max_dist_y, int bw, int max_skip, int max_iter, int n_segs, int is_cdna, int64_t n, mm128_t *a, float avg_qspan, int32_t *f, int32_t *t, int32_t *p, int32_t *v)
{
	const int blockDim = 32;
	const int maxBlockDim = 1024;
	int numBlocks = (max_iter+blockDim-1)/blockDim;
	int64_t i;
	int32_t *f_cuda, *t_cuda, *p_cuda, *v_cuda;
	mm128_t *a_cuda;
	int64_t *mem_max_j;
	int32_t *mem_max_f;
	hipError_t result;
	result = hipMalloc(&f_cuda, sizeof(int32_t)*n);
	assert(result==hipSuccess);
	result = hipMalloc(&t_cuda, sizeof(int32_t)*n);
	assert(result==hipSuccess);
	result = hipMalloc(&p_cuda, sizeof(int32_t)*n);
	assert(result==hipSuccess);
	result = hipMalloc(&v_cuda, sizeof(int32_t)*n);
	assert(result==hipSuccess);
	result = hipMalloc(&a_cuda, sizeof(mm128_t)*n);
	assert(result==hipSuccess);
	result = hipMemcpy(f_cuda, f, sizeof(int32_t)*n, hipMemcpyHostToDevice);
	assert(result==hipSuccess);
	result = hipMemcpy(t_cuda, t, sizeof(int32_t)*n, hipMemcpyHostToDevice);
	assert(result==hipSuccess);
	result = hipMemcpy(p_cuda, p, sizeof(int32_t)*n, hipMemcpyHostToDevice);
	assert(result==hipSuccess);
	result = hipMemcpy(v_cuda, v, sizeof(int32_t)*n, hipMemcpyHostToDevice);
	assert(result==hipSuccess);
	result = hipMemcpy(a_cuda, a, sizeof(mm128_t)*n, hipMemcpyHostToDevice);
	assert(result==hipSuccess);
	result = hipMalloc(&mem_max_j, sizeof(int64_t)*max_iter);
	assert(result==hipSuccess);
	result = hipMalloc(&mem_max_f, sizeof(int32_t)*max_iter);
	assert(result==hipSuccess);
	result = hipDeviceSynchronize();
	assert(result==hipSuccess);
	for (i = 0; i < n; ++i) {
		uint64_t ri = a[i].x;
		int32_t qi = (int32_t)a[i].y, q_span = a[i].y>>32&0xff; // NB: only 8 bits of span is used!!!
		int32_t sidi = (a[i].y & MM_SEED_SEG_MASK) >> MM_SEED_SEG_SHIFT;
		mm_chain_dp_fill_cuda_score<<<numBlocks, blockDim>>>(i, ri, qi, q_span, sidi, mem_max_j, mem_max_f, max_dist_x, max_dist_y, bw, max_iter, n_segs, is_cdna, a_cuda, avg_qspan, f_cuda, t_cuda, p_cuda);
		mm_chain_dp_fill_cuda_max<<<1, maxBlockDim>>>(i, mem_max_j, mem_max_f, max_iter, f_cuda, p_cuda, v_cuda);
	}
	result = hipDeviceSynchronize();
	assert(result==hipSuccess);
	result = hipMemcpy(f, f_cuda, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
	assert(result==hipSuccess);
	result = hipMemcpy(t, t_cuda, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
	assert(result==hipSuccess);
	result = hipMemcpy(p, p_cuda, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
	assert(result==hipSuccess);
	result = hipMemcpy(v, v_cuda, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
	assert(result==hipSuccess);
	hipFree(f_cuda);
	hipFree(t_cuda);
	hipFree(p_cuda);
	hipFree(v_cuda);
	hipFree(a_cuda);
	hipFree(mem_max_j);
	hipFree(mem_max_f);
}

}

