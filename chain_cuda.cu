#include <hip/hip_runtime.h>
#include "minimap.h"
#include "mmpriv.h"
#include "kalloc.h"
#include "chain_cuda_kernel.h"

extern "C" {

void mm_chain_dp_fill_cuda(int max_dist_x, int max_dist_y, int bw, int max_skip, int max_iter, int n_segs, int is_cdna, int64_t n, mm128_t *a, float avg_qspan, int32_t *f, int32_t *t, int32_t *p, int32_t *v)
{
	const int blockDim = 32;
	int numBlocks = (max_iter+blockDim-1)/blockDim;
	int real_iter = numBlocks*blockDim;
	const int n_gpu_max = 1<<24;
	int n_gpu = n<n_gpu_max?n:n_gpu_max;
	int64_t j;
	int64_t start;
	int32_t *f_cuda, *t_cuda, *p_cuda, *v_cuda;
	mm128_t *a_cuda;
	hipError_t result;
	for (uint64_t i=0;i<n;i++)
	{
		v[i] = f[i] = a[i].y>>32&0xff; // NB: only 8 bits of span is used!!!
		p[i] = -1;
	}
	result = hipMalloc(&f_cuda, sizeof(int32_t)*n_gpu);
	assert(result==hipSuccess);
	result = hipMalloc(&t_cuda, sizeof(int32_t)*n_gpu);
	assert(result==hipSuccess);
	result = hipMalloc(&p_cuda, sizeof(int32_t)*n_gpu);
	assert(result==hipSuccess);
	result = hipMalloc(&v_cuda, sizeof(int32_t)*n_gpu);
	assert(result==hipSuccess);
	result = hipMalloc(&a_cuda, sizeof(mm128_t)*n_gpu);
	assert(result==hipSuccess);
	for (start=0;start<n;)
	{
		if (start && start+real_iter>=n)
			break;
		int n_process = n-start;
		int n_copy = n_process;
		if (n_process>n_gpu)
		{
			n_process = n_gpu-real_iter;
			n_copy = n_gpu;
		}
		result = hipMemcpy(f_cuda, f+start, sizeof(int32_t)*n_copy, hipMemcpyHostToDevice);
		assert(result==hipSuccess);
		result = hipMemcpy(t_cuda, t+start, sizeof(int32_t)*n_copy, hipMemcpyHostToDevice);
		assert(result==hipSuccess);
		result = hipMemcpy(p_cuda, p+start, sizeof(int32_t)*n_copy, hipMemcpyHostToDevice);
		assert(result==hipSuccess);
		result = hipMemcpy(v_cuda, v+start, sizeof(int32_t)*n_copy, hipMemcpyHostToDevice);
		assert(result==hipSuccess);
		result = hipMemcpy(a_cuda, a+start, sizeof(mm128_t)*n_copy, hipMemcpyHostToDevice);
		assert(result==hipSuccess);
		hipDeviceSynchronize();
		assert(result==hipSuccess);
		for (j = start==0?0:start+real_iter; j < start+n_process; ++j) {
			mm_chain_dp_fill_cuda_optimized_batch<<<numBlocks, blockDim>>>(j, n, start, n_process, max_dist_x, max_dist_y, bw, max_iter, n_segs, is_cdna, a_cuda, avg_qspan, f_cuda, t_cuda, p_cuda, v_cuda);
		}
		hipDeviceSynchronize();
		assert(result==hipSuccess);
		result = hipMemcpy(f+start, f_cuda, sizeof(int32_t)*n_copy, hipMemcpyDeviceToHost);
		assert(result==hipSuccess);
		result = hipMemcpy(t+start, t_cuda, sizeof(int32_t)*n_copy, hipMemcpyDeviceToHost);
		assert(result==hipSuccess);
		result = hipMemcpy(p+start, p_cuda, sizeof(int32_t)*n_copy, hipMemcpyDeviceToHost);
		assert(result==hipSuccess);
		result = hipMemcpy(v+start, v_cuda, sizeof(int32_t)*n_copy, hipMemcpyDeviceToHost);
		assert(result==hipSuccess);
		start += n_process-real_iter;
	}
	hipFree(f_cuda);
	hipFree(t_cuda);
	hipFree(p_cuda);
	hipFree(v_cuda);
	hipFree(a_cuda);
}

}

